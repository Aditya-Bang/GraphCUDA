#include "hip/hip_runtime.h"
#include "matmul.cuh"

// Naive CUDA kernel for matrix multiplication
__global__ void matmul_kernel(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row in A and C
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column in B and C

    if (row < M && col < N) {
        float value = 0.0f;
        for (int i = 0; i < K; ++i) {
            value += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = value;
    }
}

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

template <const int BLOCKSIZE>
__global__ void sgemm_shared_mem_block(int M, int N, int K, float alpha,
                                       const float *A, const float *B,
                                       float beta, float *C) {
  // the output block that we want to compute in this threadblock
  const uint cRow = blockIdx.x;
  const uint cCol = blockIdx.y;

  // allocate buffer for current block in fast shared mem
  // shared mem is shared between all threads in a block
  __shared__ float As[BLOCKSIZE * BLOCKSIZE];
  __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

  // the inner row & col that we're accessing in this thread
  const uint threadCol = threadIdx.x % BLOCKSIZE;
  const uint threadRow = threadIdx.x / BLOCKSIZE;

  // advance pointers to the starting positions
  A += cRow * BLOCKSIZE * K;                    // row=cRow, col=0
  B += cCol * BLOCKSIZE;                        // row=0, col=cCol
  C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

  float tmp = 0.0;
  for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE) {
    // Have each thread load one of the elements in A & B
    // Make the threadCol (=threadIdx.x) the consecutive index
    // to allow global memory access coalescing
    As[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
    Bs[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];

    // block threads in this block until cache is fully populated
    __syncthreads();
    A += BLOCKSIZE;
    B += BLOCKSIZE * N;

    // execute the dotproduct on the currently cached block
    for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
      tmp += As[threadRow * BLOCKSIZE + dotIdx] *
             Bs[dotIdx * BLOCKSIZE + threadCol];
    }
    // need to sync again at the end, to avoid faster threads
    // fetching the next block into the cache before slower threads are done
    __syncthreads();
  }
  C[threadRow * N + threadCol] =
      alpha * tmp + beta * C[threadRow * N + threadCol];
}

// PyTorch wrapper
torch::Tensor matmul(torch::Tensor A, torch::Tensor B) {
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "Input tensors must be 2D");
    TORCH_CHECK(A.size(1) == B.size(0), "A.cols must match B.rows");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    // auto A_contig = A.contiguous();
    // auto B_contig = B.contiguous();

    auto C = torch::zeros({M, N}, A.options());

    float* A_ptr = A.data_ptr<float>();
    float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    // KERNEL 1 - Global Memory Coalescing
    // dim3 threads(32, 32);
    // dim3 blocks((N + threads.x - 1) / threads.x, (M + threads.y - 1) / threads.y);
    // matmul_kernel<<<blocks, threads>>>(A_ptr, B_ptr, C_ptr, M, N, K);

    // Shared Memory Cache-Blocking
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
    dim3 blockDim(32 * 32);
    hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shared_mem_block<32>),
                       hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
    sgemm_shared_mem_block<32><<<gridDim, blockDim>>>(M, N, K, 1, A_ptr, B_ptr, 0, C_ptr);



    // hipDeviceSynchronize();  // Optional: helpful for debugging

    return C;
}
